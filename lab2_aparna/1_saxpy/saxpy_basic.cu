#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "saxpy.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from dev_offsetition of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

static inline
int getBlocks(long working_set_size, int threadsPerBlock) {
  // TODO: implement and use this interface if necessary  
  return 0;
}

void 
getArrays(int size, float **xarray, float **yarray, float **resultarray) {
    /*
    *xarray = (float*) malloc(size*sizeof(float));
    *yarray = (float*) malloc(size*sizeof(float));
    *resultarray = (float*) malloc(size*sizeof(float));*/
    gpuErrchk(hipHostMalloc(xarray, size*sizeof(float)));
    gpuErrchk(hipHostMalloc(yarray, size*sizeof(float)));
    gpuErrchk(hipHostMalloc(resultarray, size*sizeof(float)));

}

void 
freeArrays(float *xarray, float *yarray, float *resultarray) {
    gpuErrchk(hipHostFree(xarray));
    gpuErrchk(hipHostFree(yarray));
    gpuErrchk(hipHostFree(resultarray));
}


void
saxpyCuda(long total_elems, float alpha, float* xarray, float* yarray, float* resultarray, int partitions) {

    const int threadsPerBlock = 512; // change this if necessary

    float *device_x;
    float *device_y;
    float *device_result;

    gpuErrchk(hipMalloc(&device_x, total_elems*sizeof(float)));
    gpuErrchk(hipMalloc(&device_y, total_elems*sizeof(float)));
    gpuErrchk(hipMalloc(&device_result, total_elems*sizeof(float)));


    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO: Compute number of thread blocks.
    // 


    double startGPUTime = CycleTimer::currentSeconds();
    //Copy to GPU
     gpuErrchk(hipMemcpy(device_x, xarray, total_elems*sizeof(float), hipMemcpyHostToDevice));
     gpuErrchk(hipMemcpy(device_y, yarray, total_elems*sizeof(float), hipMemcpyHostToDevice));
     //gpuErrchk(hipMemcpy(device_result, resultarray, total_elems*sizeof(float), hipMemcpyHostToDevice));
    double endMemCpyToDevice = CycleTimer::currentSeconds();
    timeCopyH2DAvg += (endMemCpyToDevice - startGPUTime); 

    double startKernelTime = CycleTimer::currentSeconds();

    // run saxpy_kernel on the GPU
    saxpy_kernel<<<(total_elems + threadsPerBlock - 1)/threadsPerBlock, threadsPerBlock>>>(total_elems, alpha, device_x, device_y, device_result);

    //
    // insert timer here to time only the kernel.  Since the
    // kernel will run asynchronously with the calling CPU thread, you
    // need to call hipDeviceSynchronize() before your timer to
    // ensure the kernel running on the GPU has completed.  (Otherwise
    // you will incorrectly observe that almost no time elapses!)
    //
    hipDeviceSynchronize();
    double endKernelTime = CycleTimer::currentSeconds();
    timeKernelAvg += (endKernelTime - startKernelTime);

    double endGPUTime = CycleTimer::currentSeconds();
    
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }
    
    // Copy result from GPU using hipMemcpy
    double startMemCpyToHost = CycleTimer::currentSeconds();
     gpuErrchk(hipMemcpy(resultarray, device_result, total_elems*sizeof(float), hipMemcpyDeviceToHost));
    

    // end timing after result has been copied back into host memory.
    // The time elapsed between startTime and endTime is the total
    // time to copy data to the GPU, run the kernel, and copy the
    // result back to the CPU
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    timeCopyD2HAvg += (endTime - startMemCpyToHost);
    totalTimeAvg   += overallDuration;


    // free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);

}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
