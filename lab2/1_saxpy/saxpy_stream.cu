#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "saxpy.h"

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

static inline
int getBlocks(long working_set_size, int threadsPerBlock) {
  // TODO: implement and use this interface if necessary  
}

void 
getArrays(int size, float **xarray, float **yarray, float **resultarray) {
  // TODO: implement and use this interface if necessary  
}

void 
freeArrays(float *xarray, float *yarray, float *resultarray) {
  // TODO: implement and use this interface if necessary  
}

void
saxpyCuda(long total_elems, float alpha, float* xarray, float* yarray, float* resultarray, int partitions) {

    const int threadsPerBlock = 512; // change this if necessary

    float *device_x;
    float *device_y;
    float *device_result;

    //
    // TODO: allocate device memory buffers on the GPU using
    // hipMalloc.  The started code issues warnings on build because
    // these buffers are used in the call to saxpy_kernel below
    // without being initialized.
    //
    hipMalloc(&device_x, total_elems * sizeof(float));
    hipMalloc(&device_y, total_elems * sizeof(float));
    hipMalloc(&device_result, total_elems * sizeof(float));
    
    const long NumBlocks = ((total_elems + threadsPerBlock-1)/threadsPerBlock);

    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();
    const long  size = (total_elems/partitions);
    double startGPUTime[partitions];
    double endGPUTime[partitions];
    double timeKernel[partitions];
    double endD2HTime[partitions];
    double startH2DTime[partitions];    
    hipStream_t streams[partitions];

    for (int i=0; i<100; i++) {
  
        //
        // TODO: copy input arrays to the GPU using hipMemcpy
        //
            startH2DTime[i]  = CycleTimer::currentSeconds();
       
            hipMemcpyAsync(device_x+ i * size * sizeof(float),xarray + i * size * sizeof(float), size * sizeof(float), hipMemcpyHostToDevice, streams[i]);
            hipMemcpyAsync(device_y+ i * size * sizeof(float),yarray + i * size * sizeof(float), size * sizeof(float), hipMemcpyHostToDevice, streams[i]);
             
            //
            // TODO: insert time here to begin timing only the kernel
            //
            startGPUTime[i] = CycleTimer::currentSeconds();
    
            // compute number of blocks and threads per block

            // run saxpy_kernel on the GPU
            saxpy_kernel<<<NumBlocks,threadsPerBlock,0,streams[i]>>>(total_elems,alpha,device_x,device_y,device_result);
    
            //
            // TODO: insert timer here to time only the kernel.  Since the
            // kernel will run asynchronously with the calling CPU thread, you
            // need to call hipDeviceSynchronize() before your timer to
            // ensure the kernel running on the GPU has completed.  (Otherwise
            // you will incorrectly observe that almost no time elapses!)
            //
            hipStreamSynchronize(streams[i]);
            endGPUTime[i] = CycleTimer::currentSeconds();
            timeKernel[i] = endGPUTime[i] - startGPUTime[i];
    
            hipError_t errCode = hipPeekAtLastError();
            if (errCode != hipSuccess) {
                fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
            }
    
            //
            // TODO: copy result from GPU using hipMemcpy
            //
            hipMemcpyAsync(resultarray+ i * size * sizeof(float),device_result+ i * size * sizeof(float),size * sizeof(float), hipMemcpyDeviceToHost,streams[i]);
    
            endD2HTime[i] = CycleTimer::currentSeconds();
    }

    // end timing after result has been copied back into host memory.
    // The time elapsed between startTime and endTime is the total
    // time to copy data to the GPU, run the kernel, and copy the
    // result back to the CPU
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    totalTimeAvg   += overallDuration;
    
    for(int j =0 ; j < partitions ; j++){
        timeKernelAvg  += timeKernel[j];
        timeCopyD2HAvg += endD2HTime[j] - endGPUTime[j];
        timeCopyH2DAvg += startGPUTime[j] - startH2DTime[j];
    }

    //
    // TODO free memory buffers on the GPU
    //
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
